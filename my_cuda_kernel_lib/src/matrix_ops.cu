#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "matrix_ops.h"

// CUDA kernel: C = A + B
__global__ void matrixAddKernel(const float* A, const float* B, float* C,
                                int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;

    if (row < rows && col < cols) {
        C[idx] = A[idx] + B[idx];
    }
}

// Host wrapper
void matrixAdd(const float* A, const float* B, float* C,
               int rows, int cols) {
    int size = rows * cols * sizeof(float);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((cols + block.x - 1) / block.x,
              (rows + block.y - 1) / block.y);

    matrixAddKernel<<<grid, block>>>(d_A, d_B, d_C, rows, cols);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
